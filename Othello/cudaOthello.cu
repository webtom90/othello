#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#define NDEBUG
#define nullptr 0

#define DEF __device__ __host__

#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include "Header.h"
#include "Othello.h"
#include "Watch.h"
#include "WeightsOptimizer.h"
#include "CmaEsOptimizer.h"
#include "AnyLoader.h"

template<typename T>
hipError_t cudaSetValue(UniversalLoader *dst, UniversalLoader *src, T *srcValuePos, T *value)
{
	char *dstPtr = reinterpret_cast<char *>(dst);
	char *srcPtr = reinterpret_cast<char *>(src);
	char *srcValPtr = reinterpret_cast<char *>(srcValuePos);

	char *dstValPtr = reinterpret_cast<char *>(dstPtr + (srcValPtr - srcPtr));

	return hipMemcpy(dstValPtr, value, sizeof(T), hipMemcpyHostToDevice);
}

template<typename T>
hipError_t cudaCopyValue(UniversalLoader *dst, UniversalLoader *src, T *srcValuePos)
{
	return cudaSetValue(dst, src, srcValuePos, srcValuePos);
}

template<typename T>
hipError_t cudaSetArray(UniversalLoader *dst, UniversalLoader *src, T **hostArray, int *hostSize, char *cudaData, char *hostData)
{
	int distance = reinterpret_cast<char *>(*hostArray) - hostData;
	T *pos = reinterpret_cast<T *>(cudaData + distance);
	if (*hostArray == nullptr)
		pos = nullptr;
	hipError_t cudaStatus = cudaSetValue(dst, src, hostArray, &pos);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		return cudaStatus;
	}
	cudaStatus = cudaCopyValue(dst, src, hostSize);
	if (cudaStatus != hipSuccess)
	{
        	fprintf(stderr, "hipMemcpy failed!");
        	return cudaStatus;
	}

	return cudaStatus;
}

UniversalLoader *getCudaLoader(UniversalLoader *loader)
{
	UniversalLoader *result = nullptr;
	hipError_t cudaStatus = hipMalloc((void**)&result, sizeof(UniversalLoader));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}
	cudaStatus = hipMemset(result, 0, sizeof(UniversalLoader));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemset failed!");
        goto Error;
    }
	char *data;
	cudaStatus = hipMalloc(&data, loader->dataSize * sizeof(*(loader->data)));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(data, loader->data, loader->dataSize * sizeof(*(loader->data)), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
	cudaStatus = cudaSetValue(result, loader, &loader->data, &data);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaCopyValue failed!");
        goto Error;
    }
	cudaStatus = cudaCopyValue(result, loader, &loader->dataSize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
	cudaStatus = cudaCopyValue(result, loader, &loader->type);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
	cudaStatus = cudaCopyValue(result, loader, &loader->maxTuplePerPos);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
	cudaStatus = cudaSetArray(result, loader, &loader->fields, &loader->nFields, data, loader->data);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
	cudaStatus = cudaSetArray(result, loader, &loader->weights, &loader->nWeights, data, loader->data);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
	cudaStatus = cudaSetArray(result, loader, &loader->tuples, &loader->nTuples, data, loader->data);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
	cudaStatus = cudaSetArray(result, loader, &loader->values, &loader->nValues, data, loader->data);
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	return result;
Error:

	return nullptr;
}

void removeCudaLoader(UniversalLoader *cudaLoader)
{
	UniversalLoader loader;
	UniversalLoader *normal = &loader;
	if (cudaLoader != nullptr)
	{
		int dst = reinterpret_cast<char *>(&normal->data) - reinterpret_cast<char *>(normal);
		char *pos = reinterpret_cast<char *>(cudaLoader) + dst;

		if (pos != nullptr)
			hipFree(pos);
	
		hipFree(cudaLoader);
	}
}

void getCudaLoader(AnyLoader *loader, char **data)
{
	hipError_t cudaStatus = hipMalloc((void**)data, loader->getRawDataSize());
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}
	
	cudaStatus = hipMemcpy(*data, loader->getRawData(), loader->getRawDataSize(), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	return;
Error:
	data = nullptr;
	return;
}

void removeCudaData(char **data)
{
	if (*data != nullptr)
	{
		hipFree(*data);
		*data = nullptr;
	}
}

__global__ void playGame(UniversalLoader *playerLoader, bool playerNeg, Board::EVALUATION_TYPE playerFreq, UniversalLoader **expertsLoaders, bool *expertsNeg, float expertsFreq, int nExperts, UniversalLoader *boards, Board::EVALUATION_TYPE **weights, Board::EVALUATION_TYPE *results, int gamesPerBlock, int seed)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	Rand r(seed + index);
	const int blockSize = blockDim.x;
	const int blockResultsOffset = blockIdx.x * gamesPerBlock;

	int nBoards = boards->getNValues();
	const int totalGames = nBoards * gamesPerBlock * nExperts;

	if (index == 0)
	{
		/*
		//NTuplePlayer<false, 6, 9, 3, 1> *p1 = new NTuplePlayer<false, 6, 9, 3, 1>(r.rand(), playerLoader);
		OthelloPlayer *p1 = OthelloPlayer::getPlayer(playerLoader, r.rand(), playerNeg);
		//printf("p1\n");
		NTuplePlayer<false, 6, 9, 3, 1> p2(r.rand(), expertsLoaders[0]);
		//printf("p2\n");
		p1->setRandomMoveFreq(playerFreq);
		p2.setRandomMoveFreq(expertsNeg[0]);
		//printf("f\n");

		Board b;
		//printf("Before\n");
		Othello::play(b, p1, &p2, r.rand());
		//printf("After\n");
		delete p1;*/
	}

	/*for(int i = threadIdx.x; i < gamesPerBlock; i += blockSize)
		results[i + blockResultsOffset] = 0;

	OthelloPlayer *player = OthelloPlayer::getPlayer(playerLoader, r.rand(), playerNeg);
	player->setRandomMoveFreq(playerFreq);
	__shared__ OthelloPlayer **experts;
	if (threadIdx.x == 0)
		experts = new OthelloPlayer*[nExperts];

	__syncthreads();
	for(int i = threadIdx.x; i < nExperts; i+=blockSize)
	{
		experts[i] = OthelloPlayer::getPlayer(expertsLoaders[i], r.rand(), expertsNeg[i]);
		experts[i]->setRandomMoveFreq(expertsFreq);
	}
	__syncthreads();

	Othello *othello = new Othello(player, experts, nExperts, r.rand());
	Board *tmpBoard = new Board();

	for(int i = threadIdx.x; i < totalGames; i += blockSize)
	{
		int board = i % nBoards;
		int tmp = i / nBoards;
		int game = tmp % gamesPerBlock;
		int expert = (tmp / gamesPerBlock) % nExperts;
		tmpBoard->copy(boards->getValues() + board * 64);
		Board::EVALUATION_TYPE result = othello->play(*tmpBoard, 0, expert + 1).item1;
		result = (1 - result) / (nBoards * nExperts);
		atomicAdd(results + blockResultsOffset + game, result);
	}

	delete tmpBoard;
	delete othello;

	__syncthreads();
	for(int i = threadIdx.x; i < nExperts; i+=blockSize)
		delete experts[i];
	__syncthreads();
	if (threadIdx.x == 0)
		delete[] experts;
	delete player;*/
}

class A
{
public:
	DEF A()
	{
		a = 0;
		printf("A()\t%d\n", a);
	}

	DEF A(const A &a)
	{
		this->a = a.a;
		printf("A(const A &a)\t%d\n", a);
	}

	int a;
};

DEF void f(char *data, unsigned dataSize)
{
	printf("Start f\n");

	PlayerLoader *loader = PlayerLoader::getLoader(data, dataSize, false);
	auto p1 = loader->getPlayer(0, false);
	auto p2 = loader->getPlayer(0, false);
	p1->setRandomMoveFreq(0);
	p2->setRandomMoveFreq(0);
	OthelloPlayer *players[] = { p1, p2 };
	//Othello oth(players, 2, 0);
	Othello *oth = new Othello(players, 2, 0);
	float sum = 0;
	int N = 1;

	for(int i = 0; i < N; i++)
		sum += oth->playDouble(0, 1).item1;
	
	printf("Result: %f\n", sum / N);

	delete oth;
	delete p2;
	delete p1;
	delete loader;
	printf("End f\n");
}

__global__ void playGame(char *data, unsigned dataSize)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	//printf("Start %d\n", index);

	//if (index == 0)
	{
		f(data, dataSize);
	}
}

/*class GPURunner : public GameRunner
{
protected:
	Rand rand;
	int nBlocks;
	int nThreads;
	int nGames;

	Configuration *conf;

	// Device player loader.
	UniversalLoader *cuda_pLoader;
	// Host array of device experts loaders;
	UniversalLoader **cuda_eLoaders;
	// Device array of device experts loaders.
	UniversalLoader **cudaArr_eLoaders;

	UniversalLoader *cuda_boards;
	
	bool pNeg;
	bool *eNeg;
	bool *cuda_eNeg;

	float pFreq;
	float eFreq;

	Board::EVALUATION_TYPE **cuda_weights;
	Board::EVALUATION_TYPE **cudaArr_weights;

	Board::EVALUATION_TYPE *cuda_result;

	void clear()
	{
		if (conf == nullptr)
			return;
		
		int nExperts = conf->getNExperts();
		if (cuda_pLoader != nullptr)
		{
			removeCudaLoader(cuda_pLoader);
			cuda_pLoader = nullptr;
		}
		if (cuda_eLoaders != nullptr)
		{
			for(int i = 0; i < nExperts; i++)
				removeCudaLoader(cuda_eLoaders[i]);
			delete[] cuda_eLoaders;
			cuda_eLoaders = nullptr;
		}
		if (cudaArr_eLoaders != nullptr)
		{
			hipFree(cudaArr_eLoaders);
			cudaArr_eLoaders = nullptr;
		}
		if (eNeg != nullptr)
		{
			delete[] eNeg;
			eNeg = nullptr;
		}
		if (eNeg != nullptr)
		{
			hipFree(cuda_eNeg);
			cuda_eNeg = nullptr;
		}
		if (cuda_boards != nullptr)
		{
			removeCudaLoader(cuda_boards);
			cuda_boards = nullptr;
		}
		if (cuda_result != nullptr)
		{
			hipFree(cuda_result);
			cuda_result = nullptr;
		}
		if (cuda_weights != nullptr)
		{
			for(int i = 0; i < conf->getPlayer()->getNWeights(); i++)
			{
				hipFree(cuda_weights[i]);
			}
			delete[] cuda_weights;
			cuda_weights = nullptr;
		}
		if (cudaArr_weights != nullptr)
		{
			hipFree(cudaArr_weights);
			cudaArr_weights = nullptr;
		}
	}

	virtual void _setPlayerFreq(float freq)
	{
		pFreq = freq;
	}

	virtual void _setExpertsFreq(float freq)
	{
		eFreq = freq;
	}
public:
	GPURunner(int nBlocks, int nThreads, int gamesPerBlock, int seed) :
		rand(seed),
		nBlocks(nBlocks),
		nThreads(nThreads),
		nGames(gamesPerBlock),
		conf(nullptr),
		cuda_pLoader(nullptr),
		cudaArr_eLoaders(nullptr),
		cuda_eLoaders(nullptr),
		pNeg(false),
		eNeg(nullptr),
		cuda_eNeg(nullptr),
		pFreq(0.0f),
		eFreq(0.0f),
		cuda_boards(nullptr),
		cudaArr_weights(nullptr),
		cuda_weights(nullptr),
		cuda_result(nullptr)
	{
	}

	virtual ~GPURunner()
	{
		clear();
	}

	bool init(OptimizerConfiguration *conf)
	{
		clear();

		this->conf = conf;

		int nPlayers = nBlocks * nThreads;

		{
			hipError_t result = hipDeviceSetLimit(hipLimitMallocHeapSize, nPlayers * 512 * 1024);
			if (result != hipSuccess)
			{
				printf("Cannot allocate memory!\n");
				return false;
			}
		}

		int nExperts = conf->getNExperts();
		cuda_pLoader = getCudaLoader(conf->getPlayer());
		
		cuda_eLoaders = new UniversalLoader *[nExperts];
		for(int i = 0; i < nExperts; i++)
			cuda_eLoaders[i] = getCudaLoader(conf->getExperts()[i]);
		bool correct = hipMalloc(&cudaArr_eLoaders, nExperts * sizeof(UniversalLoader *)) == hipSuccess;
		correct &= hipMemcpy(cudaArr_eLoaders, cuda_eLoaders, nExperts * sizeof(UniversalLoader *), hipMemcpyHostToDevice) == hipSuccess;

		pNeg = conf->getPlayerNeg();
		eNeg = new bool[nExperts];

		for(int i = 0; i < nExperts; i++)
			eNeg[i] = conf->getExpertsNeg()[i];

		correct &= hipMalloc(&cuda_eNeg, nExperts * sizeof(bool)) == hipSuccess;
		correct &= hipMemcpy(cuda_eNeg, eNeg, nExperts * sizeof(bool), hipMemcpyHostToDevice) == hipSuccess;

		cuda_weights = new Board::EVALUATION_TYPE *[getPreferedNWeights()];
		int nWeights = conf->getPlayer()->getNWeights();
		for(int i = 0; i < getPreferedNWeights(); i++)
		{
			correct &= hipMalloc(&cuda_weights[i], nWeights * sizeof(Board::EVALUATION_TYPE)) == hipSuccess;
		}
		correct &= hipMalloc(&cudaArr_weights, getPreferedNWeights() * sizeof(Board::EVALUATION_TYPE *)) == hipSuccess;
		correct &= hipMemcpy(cudaArr_weights, cuda_weights, getPreferedNWeights() * sizeof(Board::EVALUATION_TYPE *), hipMemcpyHostToDevice) == hipSuccess;

		correct &= hipMalloc(&cuda_result, getPreferedNWeights() * sizeof(Board::EVALUATION_TYPE)) == hipSuccess;

		cuda_boards = getCudaLoader(conf->getBoardsConf());

		if (!correct)
			clear();

		return correct;
	}

	bool run(Board::EVALUATION_TYPE *const*weights, int nWeights, Board::EVALUATION_TYPE *results)
	{
		if (nWeights != getPreferedNWeights() && nWeights != getNPlayers())
		{
			printf("Invalid number of weights (%d instead of %d)\n", nWeights, getPreferedNWeights());
			return false;
		}

		int wSize = conf->getPlayer()->getNWeights();
		bool correct = true;
		for(int i = 0; i < nWeights && correct; i++)
		{
			if (weights[i] != nullptr)
			{
				hipError_t re = hipMemcpy(cuda_weights[i], weights[i], wSize * sizeof(Board::EVALUATION_TYPE), hipMemcpyHostToDevice);
				correct &= re == hipSuccess;
			}
		}

		if (!correct)
		{
			printf("Copying weights failed %s\n", hipGetErrorString(hipGetLastError()));
			return false;
		}

		int nGames = nWeights / (nBlocks);
		if (nGames * nBlocks != nWeights)
		{
			printf("Wrong number of weights (%d instead of mul of %dx%d)\n", nWeights, nBlocks, nThreads);
			getchar();
			return false;
		}
		dim3 blocks(nBlocks);
		dim3 threads(nThreads);

		playGame<<<blocks, threads>>>(cuda_pLoader, pNeg, pFreq, cudaArr_eLoaders, cuda_eNeg, eFreq, conf->getNExperts(), cuda_boards, cudaArr_weights, cuda_result, nGames, rand.rand());

		correct = hipGetLastError() == hipSuccess;
		if (!correct)
		{
			fprintf(stderr, "playGame launch failed: %s\n", hipGetErrorString(hipGetLastError()));
			return false;
		}
    
		correct = hipDeviceSynchronize() == hipSuccess;
		if (!correct)
		{
			fprintf(stderr, "hipDeviceSynchronize returned error code %d (%s) after launching addKernel!\n", hipGetLastError(), hipGetErrorString(hipGetLastError()));
			getchar();
			return false;
		}

		correct = hipMemcpy(results, cuda_result, nWeights * sizeof(Board::EVALUATION_TYPE), hipMemcpyDeviceToHost) == hipSuccess;
		if (!correct)
		{
			printf("Copying results failed\n");
			return false;
		}

		//printf("Correct so far\n");
		return true;
	}

	int getNPlayers()
	{
		return nBlocks;
	}

	int getPreferedNWeights()
	{
		return nBlocks * nGames;
	}

	int getMinimumNWeights()
	{
		return nBlocks;
	}
};*/

int main(int argc, char **argv)
{
	Watch<float> watch;
	{
		hipError_t cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
			return -1;
		}
		AnyLoader *loader = AnyLoader::getLoader("Valid\\SzubertJaskowskiKrawiec2013CTDL.oth");
		
		watch.start();
		f(loader->getRawData(), loader->getRawDataSize());
		watch.stop();
		printf("Done after %f\n", watch());
		/*getchar();
		return;*/

		char *data;
		getCudaLoader(loader, &data);
		
		dim3 blocks(1);
		dim3 threads(1);

		watch.start();
		playGame<<<blocks, threads>>>(data, loader->getRawDataSize());
		
		bool correct = hipGetLastError() == hipSuccess;
		if (!correct)
		{
			fprintf(stderr, "playGame launch failed: %s\n", hipGetErrorString(hipGetLastError()));
		}

		correct = hipDeviceSynchronize() == hipSuccess;
		if (!correct)
		{
			fprintf(stderr, "hipDeviceSynchronize returned error code %d (%s) after launching addKernel!\n", hipGetLastError(), hipGetErrorString(hipGetLastError()));
			getchar();
			return false;
		}

		watch.stop();
		printf("Done after %.2fs\n", watch());
		
		removeCudaData(&data);

		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipDeviceReset failed!\n");
		}

		getchar();

		return 0;
	}
}
