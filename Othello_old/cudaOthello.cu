#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#define NDEBUG

#define DEF __device__ __host__

#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include "Othello.h"
#include "Watch.h"

hipError_t runGame(Watch<float> &w, int *c, float *res1, float *res2, const unsigned height, const unsigned width, int time, int size);

template<typename T>
__device__ void r(int *c, float *res1, float *res2, int n)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int value = c[index];
	T p1(value);
	T p2(value + 583);
	int counter = 0;
	float r1 = 0;
	float r2 = 0;
	for(int i = 0; i < n; i++)
	{
		Tuple<Board::EVALUATION_TYPE, Board::EVALUATION_TYPE> result = Othello::playDouble(&p1, &p2);
		r1 += result.item1;
		r2 += result.item2;
		counter++;
	}
	c[index] = counter;
	res1[index] = r1;
	res2[index] = r2;
}

__device__ void run(int *c, float *res1, float *res2, int n, int size)
{
	switch (size)
	{
		case 0:
			r<CpuPlayer1>(c, res1, res2, n);
			break;
		case 1:
			r<TuplePlayer<1> >(c, res1, res2, n);
			break;
		case 2:
			r<TuplePlayer<2> >(c, res1, res2, n);
			break;
		case 3:
			r<TuplePlayer<3> >(c, res1, res2, n);
			break;
		case 4:
			r<TuplePlayer<4> >(c, res1, res2, n);
			break;
	}
}

__global__ void runOnGPU(int *array, float *res1, float *res2, int time, int size)
{
	run(array, res1, res2, time, size);
}

int main(int argc, char **argv)
{
#define N_ITER 8
	int count = 1;
	int seed = 0;
	int size = 1;
	if (argc > 1)
	{
		seed = atoi(argv[1]);
	}
	if (argc > 2)
	{
		size = atoi(argv[2]);
	}
	srand(0);
	Vector<double, N_ITER-1> times;
	for(int t = 0; t < N_ITER; t++)
	{
	Watch<float> w;
	const int width = 64;
	const int height = 21 * 1;
	const int arraySize = width * height;
	int c[arraySize];
	for(int i = 0; i < arraySize; i++)
		c[i] = seed;
	float r1[arraySize] = { 0 };
	float r2[arraySize] = { 0 };

	// Add vectors in parallel.
	hipError_t cudaStatus = runGame(w, c, r1, r2, height, width, count, size);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addWithCuda failed!");
		getchar();
		return 1;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceReset failed!");
		getchar();
		return 1;
	}
	
	int totalCounter = 0;
	float w1 = 0;
	float w2 = 0;
	for(int i = 0; i < arraySize; i++)
	{
		if (c[i] <= 0)
		{
			printf("Zly wynik: %d (%d)\n", i, c[i]);
			break;
		}
		totalCounter += c[i];
		w1 += r1[i];
		w2 += r2[i];
	}

	if (t > 0)
		times.add(totalCounter/w());
	}
	double avg(0), dev(0);
	for(int i = 0; i < times.size(); i++)
		avg += times[i] / times.size();
	for(int i = 0; i < times.size(); i++)
		dev += (times[i] - avg) * (times[i] - avg);
	dev = sqrt(dev / times.size());
	printf("GPS: %.1f, %.3f\n", avg, dev);

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t runGame(Watch<float> &w, int *c, float *res1, float *res2, const unsigned height, const unsigned width, int time, int N)
{
    const int size = height * width;
    int *dev_c = 0;
    float *dev_r1 = 0;
    float *dev_r2 = 0;
    dim3 blockGridRows(height);
    dim3 threadBlockRows(width);
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

	// Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_r1, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_r2, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_c, c, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    
    cudaStatus = hipFuncSetCacheConfig(reinterpret_cast<const void*>(runOnGPU), hipFuncCachePreferL1);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSetCacheConfig launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    w.start();
    // Launch a kernel on the GPU with one thread for each element.
    runOnGPU<<<blockGridRows, threadBlockRows>>>(dev_c, dev_r1, dev_r2, time, N);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "runOnGPU launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching runOnGPU!\n", cudaStatus);
        goto Error;
    }
    w.stop();

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(res1, dev_r1, size * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(res2, dev_r2, size * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_r1);
    hipFree(dev_r2);
    
    return cudaStatus;
}
